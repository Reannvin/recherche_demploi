#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixMul(float* A, float* B, float* C, int M, int N, int K){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < M && col < N){
        float sum = 0.0f;
        for (int i = 0;i < K; i ++){
            sum += A[row * K + i] * B[i * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main(){
    const int M = 1024, N = 1024, K = 1024;
    const int SIZE = M * N * sizeof(float);
    
    float* h_A = (float*)malloc(M * K * sizeof(float));
    float* h_B = (float*)malloc(K * N * sizeof(float));
    float* h_C = (float*)malloc(M * N * sizeof(float));

    float* d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * M * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16,16);
    dim3 numBlocks(N / threadsPerBlock.x, M/threadsPerBlock.y);
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, M, N, K);

    hipMemcpy(h_C, d_C,M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}