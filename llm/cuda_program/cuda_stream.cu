#include <hip/hip_runtime.h>
#include <iostream>

hipStream_t stream1, stream2;
hipStreamCreate(&stream1);
hipStreamCreate(&stream2);

float* d_a, *d_b, *d_c;
hipMalloc(&d_a,...);
hipMalloc(&d_b,...);
hipMalloc(&d_c,...);

// Exec kernal in Stream1
kernal<<<...,0,stream1>>>(d_a, d_b, d_c);

// Exec transfer from CPU to GPU in Stream2
hipMemcpyAsync(d_a,...,hipMemcpyHostToDevice,stream2);

// Exec transfer from GPU to CPU in Stream1 after Stream2 done.
hipMemcpyAsync(d_b.d_a,...,hipMemcpyHostToDevice,stream1);

// Wait stream1 finished its kernal.
hipStreamSynchronize(stream1);

