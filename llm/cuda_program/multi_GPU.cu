#include <hip/hip_runtime.h>
#include <iostream>

// 1.Device Management
int num_devices;
hipGetDeviceCount(&num_devices);

for(int i = 0; i<num_devices; i++){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("GPU %d: %s\n,i,prop.name");
}

hipSetDevice(0);

// 2.Multi-GPU Kernal Launch
hipStream_t streams[NUM_GPUS];
for(int i = 0; i<NUM_GPUS; i++){
    hipSetDevice(i);
    hipStreamCreate(&streams[i]);
    kernal<<...,0, streams[i]<>>>(args)
}

for(int i = 0; i<NUM_GPUS; i++){
    hipSetDevice(i);
    hipStreamSynchronize(streams[i]);
}

// 3.Multi-GPU Communication
for(int i=0; i<num_devices;i++){
    hipSetDevice(i);
    for(int j = 0;j<num_devices; j++){
        int can_access;
        if(i != j){
            hipDeviceCanAccessPeer(&can_access, i, j);
            printf("GPU%d -> GPU%d: %s\n", i, j, can_access ? "Yes" : "No");
        }
    }
}

hipDeviceEnablePeerAccess(1, 0);
...//

hipMemcpyPeerAsync(...);