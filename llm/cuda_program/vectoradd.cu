#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(float* a, float* b, float* c, int n)
{
    int idx = blockDim.x * blockIdx.x + threadId.x;
    if (idx < n)
        c[idx] = a[idx] + b[idx];
}

int main()
{
    int N = 1 << 20;
    size_t bytes = N * sizeof(float);
    
    // Alocate host vectors
    float *h_a, *h_b, *h_c;
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    // Initialize host vector
    float* d_a, *d_b, *d_c
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1)  / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(h_c,d_c.bytes, cudaMemcpyDevicetoHost)

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);
}